#include "hip/hip_runtime.h"
#include <stdio.h>
#include <hip/hip_runtime.h>
#define STB_IMAGE_IMPLEMENTATION
#define STB_IMAGE_WRITE_IMPLEMENTATION

#include "stb_image.h"
#include "stb_image_write.h"


// CUDA Kernel to convert RGB to Grayscale
__global__ void rgb_to_grayscale(unsigned char* d_input, unsigned char* d_output, int width, int height, int channels) {
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;
    
    if (x < width && y < height) {
        int idx = (y * width + x) * channels;
        unsigned char r = d_input[idx];
        unsigned char g = d_input[idx + 1];
        unsigned char b = d_input[idx + 2];
        
        // Grayscale formula: 0.299*R + 0.587*G + 0.114*B
        unsigned char gray = static_cast<unsigned char>(0.299f * r + 0.587f * g + 0.114f * b);
        
        d_output[y * width + x] = gray;
    }
}

int main() {
    // Load image using stb_image
    int width, height, channels;
    unsigned char* h_input = stbi_load("D:/2023 all FILES/IMG_20210309_124347.jpg", &width, &height, &channels, 3);  // Force 3 channels (RGB)
    if (!h_input) {
        printf("Error: Failed to load image!\n");
        return -1;
    }

    size_t img_size = width * height * channels;
    size_t gray_size = width * height;

    // Allocate device memory
    unsigned char *d_input, *d_output;
    hipMalloc((void**)&d_input, img_size);
    hipMalloc((void**)&d_output, gray_size);

    // Copy data to device
    hipMemcpy(d_input, h_input, img_size, hipMemcpyHostToDevice);

    // Launch CUDA kernel
    dim3 blockSize(16, 16);
    dim3 gridSize((width + blockSize.x - 1) / blockSize.x, (height + blockSize.y - 1) / blockSize.y);
    rgb_to_grayscale<<<gridSize, blockSize>>>(d_input, d_output, width, height, channels);
    
    // Copy result back to host
    unsigned char* h_output = (unsigned char*)malloc(gray_size);
    hipMemcpy(h_output, d_output, gray_size, hipMemcpyDeviceToHost);

    // Save grayscale image using stb_image_write
    stbi_write_jpg("output.jpg", width, height, 1, h_output, 100);

    // Free memory
    hipFree(d_input);
    hipFree(d_output);
    free(h_output);
    stbi_image_free(h_input);

    printf("Grayscale conversion completed. Check output.jpg\n");
    return 0;
}


// you have to save these two files in the local folder   stb_image.h and stb_image_write.h  save them from official github repo
