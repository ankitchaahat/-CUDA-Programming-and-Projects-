
#include <hip/hip_runtime.h>
#include<iostream>

__global__ void hello_from_gpu()    // kernel code
{
  printf("Hello from GPU!\n");     // this is what we need to print  , this is on gpu
}

int main()                         // entry point
{
  hello_from_gpu<<<1, 1>>>();      //  threads per block, blocks per grid

  hipDeviceSynchronize();         // wait for all the threads to complete their work 

  return 0;
}

// to compile the program   nvcc hello_from_gpu.cu -o hello_from_gpu.exe               .\hello_from_gpu.exe
