#include<stdio.h>
#include<hip/hip_runtime.h>


__global__ void add(int *a, int *b, int *c, int n)
{
    int index = threadIdx.x + blockIdx.x * blockDim.x;
    if(index < n)
    {
        c[index] = a[index] + b[index];
    }
    
}
int main()
{
    int n = 1024;

    int *h_a, *h_b, *h_c;
    int *d_a, *d_b, *d_c;
    size_t  size = n * sizeof(int);

    // Allocate host memory
    h_a = (int *)malloc(size);
    h_b = (int *)malloc(size);
    h_c = (int *)malloc(size);

    // initialize input vectors
    for(int i = 0; i < n; i++)
    {
        h_a[i] = i;
        h_b[i] = i * 2;
    }

    // Allocate device memory
    hipMalloc((void **)&d_a, size);
    hipMalloc((void **)&d_b, size);
    hipMalloc((void **)&d_c, size);

    // copy vectors from host to device
    hipMemcpy(d_a, h_a, size, hipMemcpyHostToDevice);
    hipMemcpy(d_b, h_b, size, hipMemcpyHostToDevice);

    // launch kernel
    int threadsPerBlock(256);
    int blocksPerGrid((n + threadsPerBlock - 1) / threadsPerBlock);
    add<<<blocksPerGrid, threadsPerBlock>>>(d_a, d_b, d_c, n);

    // copy result from device to host
    hipMemcpy(h_c, d_c, size, hipMemcpyDeviceToHost);

    // print result
    for(int i = 0 ; i < n; i++)
    {
        printf(" %d + %d = %d\n", h_a[i], h_b[i], h_c[i]);
    }

    // free device memory
    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);

    // free host memory
    free(h_a);
    free(h_b);
    free(h_c);
}
