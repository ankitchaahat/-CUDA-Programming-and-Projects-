#include "hip/hip_runtime.h"
#include<stdio.h>
#include<stdlib.h>
#include<hip/hip_runtime.h>
#include<chrono>

__global__ void add_gpu(int *a, int *b, int *c, int n)
{
    int index = threadIdx.x + blockIdx.x * blockDim.x;

    if(index < n)
    {
        c[index] = b[index] + a[index];
    }
}

void add_cpu(int *a, int *b, int *c, int n)
{
    for(int i = 0; i < n; i++)
    {
        c[i] = a[i] + b[i];

    }
}

int main()
{
    int n = 1 << 20; // 2^20 = 1048576
    int size = n * sizeof(int);

    int *h_a, int *h_b, int *h_c;
    int *d_a, int *d_b, int *d_c;
    
    h_a = (int *)malloc(size);
    h_b = (int *)malloc(size);
    h_c = (int *)malloc(size);

    // initialize arrays
    for(int i = 0; i < n; i++)
    {
        h_a[i] = i;
        h_b[i] = i*2;
    }

    // CPU timing
    auto start_cpu = std::chrono::high_resolution_clock::now();
    add_cpu(h_a, h_b, h_c, n);
    auto end_cpu = std::chrono::high_resolution_clock::now();
    std::chrono::duration<double> cpu_duration = end_cpu - start_cpu;

    // allocate device memory
    hipMalloc((void **)&d_a, size);
    hipMalloc((void **)&d_b, size);
    hipMalloc((void **)&d_c, size);

    // copy data from host to device
    hipMemcpy(d_a, h_a, size, hipMemcpyHostToDevice);
    hipMemcpy(d_b, h_b, size, hipMemcpyHostToDevice);

    // GPU timing
    hipEvent_t start_gpu, stop_gpu;
    hipEventCreate(&start_gpu);
    hipEventCreate(&stop_gpu);
    hipEventRecord(start_gpu);

    int threadsPerBlock = 256;
    int blocksPerGrid = (n + threadsPerBlock - 1) / threadsPerBlock;
    add_gpu<<<blocksPerGrid, threadsPerBlock>>>(d_a, d_b, d_c, n);

    hipEventRecord(stop_gpu);
    hipEventSynchronize(stop_gpu);

    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start_gpu, stop_gpu);
    
     // Copy result back to host
     hipMemcpy(h_c, d_c, size, hipMemcpyDeviceToHost);

    // --- Timing Output ---
    printf("CPU time: %.4f seconds\n", cpu_duration.count());
    printf("GPU time: %.4f milliseconds\n", milliseconds);
    printf("Speedup: %.2fx\n", (cpu_duration.count() * 1000) / milliseconds);

    // Cleanup
    free(h_a);
    free(h_b);
    free(h_c);
    free(h_c);
    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);

    return 0;


}
