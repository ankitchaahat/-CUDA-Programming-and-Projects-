#include<hip/hip_runtime.h>
#include<iostream>
#include<chrono>

#define N (1 << 20) // 2^20 = 1048576
#define NUM_STREAMS 4


__global__ void intArray(int *arr, int value, int size){
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
    if(idx < size){
        arr[idx] = value;
    }
}

int main()
{
    int *h_data = new int[N];
    int *d_data;

    hipMalloc((void **)&d_data, N*sizeof(int));

    // stream creation
    hipStream_t streams[NUM_STREAMS];
    for(int i = 0; i < NUM_STREAMS; i++){
        hipStreamCreate(&streams[i]);
    }

    int chunksize = N / NUM_STREAMS;
    int blocksize = 256;
    int gridsize = (chunksize + blocksize -1 ) / blocksize;

    // launch kernel in parallel stremas
    for(int i = 0; i < NUM_STREAMS; i++){
        int offset = i * chunksize;
        intArray<<<gridsize, blocksize, 0, streams[i]>>>(d_data + offset, i+1, chunksize);
    }

    // copy data from device to host
    hipMemcpy(h_data, d_data, N*sizeof(int), hipMemcpyDeviceToHost);

    // wait for all streams to finish
    for(int i = 0; i < NUM_STREAMS; i++){
        hipStreamSynchronize(streams[i]);
        hipStreamDestroy(streams[i]);
    }

    // validate results
    bool correct = true;
    for(int i = 0; i < N; i++){
        if(h_data[i] != (i / chunksize) + 1){
            correct = false;
            break;
        }
    }
    if(correct){
        std::cout << "Data is correct!" << std::endl;
    } else {
        std::cout << "Data is incorrect!" << std::endl;
    }

    // print first 10 elements of the result
    std::cout << "First 10 elements of the result: " << std::endl;
    for(int i = 0; i < 10; i++){
        std::cout << h_data[i] << " ";
    }

    // free memory
    delete[] h_data;
    hipFree(d_data);
    return 0;

    
}
    
