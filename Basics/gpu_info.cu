#include <iostream>
#include <hip/hip_runtime.h>

int main() {
    int deviceCount;
    hipGetDeviceCount(&deviceCount);

    if (deviceCount == 0) {
        std::cout << "No CUDA devices found.\n";
        return 1;
    }

    for (int i = 0; i < deviceCount; ++i) {
        hipDeviceProp_t prop;
        hipGetDeviceProperties(&prop, i);

        std::cout << "=== Device " << i << " ===\n";
        std::cout << "Name: " << prop.name << "\n";
        std::cout << "Compute Capability: " << prop.major << "." << prop.minor << "\n";
        std::cout << "Multiprocessors (SMs): " << prop.multiProcessorCount << "\n";
        std::cout << "Total Global Memory: " << (prop.totalGlobalMem >> 20) << " MB\n";
        std::cout << "Shared Memory per Block: " << (prop.sharedMemPerBlock >> 10) << " KB\n";
        std::cout << "Max Threads per Block: " << prop.maxThreadsPerBlock << "\n";
        std::cout << "Max Threads per SM: " << prop.maxThreadsPerMultiProcessor << "\n";
        std::cout << "Warp Size: " << prop.warpSize << "\n";

        std::cout << "Max Grid Size: [" << prop.maxGridSize[0] << ", "
                  << prop.maxGridSize[1] << ", " << prop.maxGridSize[2] << "]\n";
        std::cout << "Max Block Dimensions: [" << prop.maxThreadsDim[0] << ", "
                  << prop.maxThreadsDim[1] << ", " << prop.maxThreadsDim[2] << "]\n";

        std::cout << "----------------------------------------\n";
    }

    return 0;
}
