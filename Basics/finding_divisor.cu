#include<stdio.h>
#include<hip/hip_runtime.h>

__global__ void finding_divisor(int *input, int *output, int n) {
    __shared__ int shared_memory[256];  

    int tid = threadIdx.x;
    int idx = blockIdx.x * blockDim.x + threadIdx.x;

    if (idx < n) {
        shared_memory[tid] = input[idx];  
    }
    __syncthreads();

    if (idx < n) {
        int num = shared_memory[tid];  
        int count = 0;  

        for (int i = 1; i <= num; i++) {
            if (num % i == 0) {
                output[idx * 10 + count] = i;  
                count++;
            }
        }
        output[idx * 10 + count] = -1; // End marker
    }
}

int main() {
    int n = 256;
    int *h_input, *h_output;
    int *d_input, *d_output;

    h_input = (int *)malloc(n * sizeof(int));
    h_output = (int *)malloc(n * 10 * sizeof(int));  // Max 10 divisors per number

    for (int i = 0; i < n; i++) {
        h_input[i] = i + 1;
    }

    hipMalloc((void **)&d_input, n * sizeof(int));
    hipMalloc((void **)&d_output, n * 10 * sizeof(int));

    hipMemcpy(d_input, h_input, n * sizeof(int), hipMemcpyHostToDevice);

    int threads_per_block = 256;
    int numBlocks = (n + threads_per_block - 1) / threads_per_block;

    finding_divisor<<<numBlocks, threads_per_block>>>(d_input, d_output, n);

    hipMemcpy(h_output, d_output, n * 10 * sizeof(int), hipMemcpyDeviceToHost);

    // Print divisors
    for (int i = 0; i < n; i++) {
        printf("Divisors of %d: ", h_input[i]);
        for (int j = 0; j < 10; j++) {
            if (h_output[i * 10 + j] == -1) break;
            printf("%d ", h_output[i * 10 + j]);
        }
        printf("\n");
    }

    free(h_input);
    free(h_output);
    hipFree(d_input);
    hipFree(d_output);

    return 0;
}

