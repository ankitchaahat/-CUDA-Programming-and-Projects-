// Sum of array elements using parallel reduction

#include <iostream>
#include<hip/hip_runtime.h>
#define N 256

__global__ void parallel_reduction(float *input, float *output,  int n)
{
    __shared__ float shared_memory[N];
    
    int tid = threadIdx.x;
    int idx = blockIdx.x * blockDim.x + threadIdx.x;

    if(tid < N)
    {
        shared_memory[tid] = input[idx];
    }
    __syncthreads();

    // parallel reduction
    for(int stride = blockDim.x / 2; stride > 0; stride /= 2)
    {
        if(tid < stride)
        {
            shared_memory[tid] += shared_memory[tid + stride];
        }
        __syncthreads();
    }

    if(tid == 0)
    {
        output[blockIdx.x] = shared_memory[0];  // saving all the sum of each block in the output array
    }
}

int main()
{
    int n = N;
    float *h_input, *h_output;
    float *d_input, *d_output;
    float sum = 0;

    h_input = (float *)malloc(n * sizeof(float));
    h_output = (float *)malloc(n * sizeof(float));

    for(int i = 0; i < n; i++)
    {
        h_input[i] = i;
    }

    hipMalloc((void **)&d_input, n * sizeof(float));
    hipMalloc((void **)&d_output, n * sizeof(float));

    hipMemcpy(d_input, h_input, n * sizeof(float),  hipMemcpyHostToDevice);

    int threads_per_block = n;
    int numBlocks = (N + threads_per_block - 1) / threads_per_block;

    parallel_reduction<<<numBlocks, threads_per_block>>>(d_input, d_output, n);

    hipMemcpy(h_output, d_output, n * sizeof(float), hipMemcpyDeviceToHost);

    for(int i = 0; i < n; i++)
    {
        sum += h_output[i];
    }

    std::cout << "Sum of array elements: " << sum << std::endl;

    free(h_input);
    free(h_output);
    hipFree(d_input);
    hipFree(d_output);

    return 0;
}
